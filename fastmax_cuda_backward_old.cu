#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <chrono>

// kernel = a0 + a1x + a2x^2
// __device__ float a0 = 1.0;
// __device__ float a1 = 1.166666;
// __device__ float a2 = 0.145833;
// __device__ float a22 = 0.145833*2;
// // -lim^2 <= q.k <= lim^2
// __device__ float lim = 2;

// // // kernel = a0 + a1x + a2x^2
// __device__ float a0 = 1.0;
// __device__ float a1 = 1.0;
// __device__ float a2 = 0.5;
// __device__ float a22 = 0.5*2;
// // -lim^2 <= q.k <= lim^2
// __device__ float lim = 1;

namespace {
__global__
void calc_gradq_unmasked(torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> q, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> k, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> v, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> o, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> gradq, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> grad_output, int bh, int nq, int nk, int d, float a1, float a22, int p,int bhratio){
  extern __shared__ float s[];
  const int m = threadIdx.x;
  const int i = blockIdx.x;
  int ikv;
  float tv, t;
  int loc1, loc2;
  float tr[32];
  int sz = min(32,d);
  if(m < d && i < bh){
    ikv = i/bhratio;
    // UNMASKED PART ////////////////////////////
        // calc q 0 
    for(int oo = 0; oo < d/sz; ++oo){
      for(int outer = 0; outer < sz; ++outer) tr[outer] = 0;
      for(int l = 0; l < nk; l++){
        tv = k[ikv][l][m];
        s[m] = v[ikv][l][m];
        __syncthreads();
        for(int outer = 0; outer < sz; ++outer){
          tr[outer] += a1*tv*s[oo*sz+outer];
        }
      }
      for(int l = 0; l < nq; l++){
        t = 0;
        s[d+m] = grad_output[i][l][m];
        __syncthreads();
        for(int outer = 0; outer < sz; ++outer){
          t += tr[outer]*s[d+oo*sz+outer];
        }
        gradq[i][l][m] += t;
      }

      for(int outer = 0; outer < sz; ++outer) tr[outer] = 0;
      for(int l = 0; l < nk; l++){
        tv = k[ikv][l][m];
        for(int outer = 0; outer < sz; ++outer){
          tr[outer] += a1*tv;
        }
      }
      for(int l = 0; l < nq; l++){
        t = 0;
        s[m] = o[i][l][m];
        s[d+m] = grad_output[i][l][m];
        __syncthreads();
        for(int outer = 0; outer < sz; ++outer){
          int ooo = oo*sz+outer;
          t += tr[outer]*s[ooo]*s[d+ooo];
        }
        gradq[i][l][m] -= t;
      }
    }

    // // calc q 0 
    // for(int outer = 0; outer < d; ++outer){
    //   tv = 0;
    //   t = 0;
    //   for(int l = 0; l < nk; l++){
    //     tv += a1*k[l][i][m]*v[l][i][outer];
    //     t += a1*k[l][i][m];
    //   }
    //   for(int l = 0; l < nq; l++){
    //     gradq[l][i][m] += (tv - t*o[l][i][outer]) * grad_output[l][i][outer];
    //   }
    // }

    // // calc q 1
    // if(p == 2){
    //   for(int outer = 0; outer < d; ++outer){
    //     for(int rr = 0; rr < d/sz; ++rr){
    //       for(int r = 0; r < sz; ++r) tr[r]= 0;
    //       for(int l = 0; l < nk; l++){
    //         s[d+m] = k[l][i][m];
    //         tv = v[l][i][outer]*k[l][i][m];
    //         __syncthreads();          
    //         loc1 = d+rr*sz;
    //         for(int r = 0; r < sz; ++r){
    //           tr[r] += s[loc1+r]*tv;
    //         }
    //       }
    //       for(int l = 0; l < nq; l++){
    //         s[m] = q[l][i][m];
    //         __syncthreads();
    //         t = 0;
    //         loc2 = rr*sz;
    //         for(int r = 0; r < sz; ++r){
    //           t += tr[r]*s[loc2+r];
    //         }
    //         gradq[l][i][m] += a22*t*grad_output[l][i][outer];
    //       }

    //       for(int r = 0; r < sz; ++r) tr[r]= 0;
    //       for(int l = 0; l < nk; l++){
    //         s[d+m] = k[l][i][m];
    //         tv = k[l][i][m];
    //         __syncthreads();          
    //         loc1 = d+rr*sz;
    //         for(int r = 0; r < sz; ++r){
    //           tr[r] += s[loc1+r]*tv;
    //         }
    //       }
    //       for(int l = 0; l < nq; l++){
    //         s[m] = q[l][i][m];
    //         __syncthreads();
    //         t = 0;
    //         loc2 = rr*sz;
    //         for(int r = 0; r < sz; ++r){
    //           t += tr[r]*s[loc2+r];
    //         }
    //         gradq[l][i][m] -= a22*t*grad_output[l][i][outer]*o[l][i][outer];
    //       }
    //     }
    //   }
    // }
  }
}

__global__
void calc_gradq_masked(torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> q, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> k, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> v, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> o, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> gradq, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> grad_output, int bh, int nq, int nk, int d, float a1, float a22, int p,int bhratio){
  extern __shared__ float s[];
  const int m = threadIdx.x;
  const int i = blockIdx.x;
  int ikv;
  float tv, t;
  int loc1, loc2;
  float tr[32];
  int sz = min(32,d);
  if(m < d && i < bh){
    ikv = i/bhratio;
    // MASKED PART ////////////////////////////
    // calc q 0 
    for(int oo = 0; oo < d/sz; ++oo){
      for(int outer = 0; outer < sz; ++outer) tr[outer] = 0;
      for(int l = 0; l < nk-nq; l++){
        tv = k[ikv][l][m];
        s[m] = v[ikv][l][m];
        __syncthreads();
        for(int outer = 0; outer < sz; ++outer){
          tr[outer] += a1*tv*s[oo*sz+outer];
        }
      }
      for(int l = 0; l < nq; l++){
        t = 0;
        tv = k[ikv][l][m];
        s[m] = v[ikv][l][m];
        s[d+m] = grad_output[i][l][m];
        __syncthreads();
        for(int outer = 0; outer < sz; ++outer){
          int ooo = oo*sz+outer;
          tr[outer] += a1*tv*s[ooo];
          t += tr[outer]*s[d+ooo];
        }
        gradq[i][l][m] += t;
      }

      for(int outer = 0; outer < sz; ++outer) tr[outer] = 0;
      for(int l = 0; l < nk-nq; l++){
        tv = k[ikv][l][m];
        for(int outer = 0; outer < sz; ++outer){
          tr[outer] += a1*tv;
        }
      }
      for(int l = 0; l < nq; l++){
        t = 0;
        tv = k[ikv][l][m];
        s[m] = o[i][l][m];
        s[d+m] = grad_output[i][l][m];
        __syncthreads();        
        for(int outer = 0; outer < sz; ++outer){
          int ooo = oo*sz+outer;
          tr[outer] += a1*tv;
          t += tr[outer]*s[ooo]*s[d+ooo];
        }
        gradq[i][l][m] -= t;
      }
    }

    // // calc q 0 
    // for(int outer = 0; outer < d; ++outer){
    //   tv = 0;
    //   t = 0;
    //   for(int l = 0; l < nk-nq; l++){
    //     tv += a1*k[l][i][m]*v[l][i][outer];
    //     t += a1*k[l][i][m];
    //   }
    //   for(int l = 0; l < nq; l++){
    //     tv += a1*k[nk-nq+l][i][m]*v[nk-nq+l][i][outer];
    //     t += a1*k[nk-nq+l][i][m];
    //     gradq[l][i][m] += (tv - t*o[l][i][outer]) * grad_output[l][i][outer];
    //   }
    // }

    // // calc q 1
    // if(p == 2){
    //   for(int outer = 0; outer < d; ++outer){
    //     for(int rr = 0; rr < d/sz; ++rr){
    //       for(int r = 0; r < sz; ++r) tr[r]= 0;
    //       for(int l = 0; l < nk-nq; l++){
    //         s[d+m] = k[l][i][m];
    //         tv = v[l][i][outer]*k[l][i][m];
    //         __syncthreads();          
    //         loc1 = d+rr*sz;
    //         for(int r = 0; r < sz; ++r){
    //           tr[r] += s[loc1+r]*tv;
    //         }
    //       }
    //       for(int l = 0; l < nq; l++){
    //         s[d+m] = k[nk-nq+l][i][m];
    //         tv = v[nk-nq+l][i][outer]*k[nk-nq+l][i][m];
    //         s[m] = q[l][i][m];
    //         __syncthreads();          
    //         loc1 = d+rr*sz;
    //         for(int r = 0; r < sz; ++r){
    //           tr[r] += s[loc1+r]*tv;
    //         }
    //         t = 0;
    //         loc2 = rr*sz;
    //         for(int r = 0; r < sz; ++r){
    //           t += tr[r]*s[loc2+r];
    //         }
    //         gradq[l][i][m] += a22*t*grad_output[l][i][outer];
    //       }

    //       for(int r = 0; r < sz; ++r) tr[r]= 0;
    //       for(int l = 0; l < nk-nq; l++){
    //         s[d+m] = k[nk-nq+l][i][m];
    //         tv = k[l][i][m];
    //         __syncthreads();          
    //         loc1 = d+rr*sz;
    //         for(int r = 0; r < sz; ++r){
    //           tr[r] += s[loc1+r]*tv;
    //         }
    //       }
    //       for(int l = 0; l < nq; l++){
    //         s[d+m] = k[nk-nq+l][i][m];
    //         tv = k[nk-nq+l][i][m];
    //         s[m] = q[l][i][m];
    //         __syncthreads();         
    //         loc1 = d+rr*sz;
    //         for(int r = 0; r < sz; ++r){
    //           tr[r] += s[loc1+r]*tv;
    //         }
    //         t = 0;
    //         loc2 = rr*sz;
    //         for(int r = 0; r < sz; ++r){
    //           t += tr[r]*s[loc2+r];
    //         }
    //         gradq[l][i][m] -= a22*t*grad_output[l][i][outer]*o[l][i][outer];
    //       }
    //     }
    //   }
    // }
  }
}


__global__
void calc_gradk_unmasked(torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> q, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> k, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> v, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> o, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> gradk, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> grad_output, int bh, int nq, int nk, int d, float a1, float a22, int p,int bhratio){

  extern __shared__ float s[];
  const int m = threadIdx.x;
  const int i = blockIdx.x;
  int ikv;
  float tv, t;
  int loc1, loc2;
  float tr[32];
  int sz = min(32,d);
  if(m < d && i < bh){
    ikv = i/bhratio;
    // UNMASKED PART ////////////////////////////
    for(int oo = 0; oo < d/sz; ++oo){
      for(int outer = 0; outer < sz; ++outer) tr[outer] = 0;
      for(int l = 0; l < nq; l++){
        tv = q[i][l][m];
        s[m] = grad_output[i][l][m];
        __syncthreads();
        for(int outer = 0; outer < sz; ++outer){
          tr[outer] += a1*tv*s[oo*sz+outer];
        }
      }
      for(int l = 0; l < nk; l++){
        t = 0;
        s[d+m] = v[ikv][l][m];
        __syncthreads();
        for(int outer = 0; outer < sz; ++outer){
          t += tr[outer]*s[d+oo*sz+outer];
        }
        gradk[ikv][l][m] += t;
      }

      for(int outer = 0; outer < sz; ++outer) tr[outer] = 0;
      for(int l = 0; l < nq; l++){
        t = 0;
        tv = q[i][l][m];
        s[m] = grad_output[i][l][m];
        s[d+m] = o[i][l][m];
        __syncthreads();
        for(int outer = 0; outer < sz; ++outer){
          int ooo = oo*sz + outer;
          tr[outer] += a1*s[d+ooo]*tv*s[ooo];
        }
      }

      for(int l = 0; l < nk; l++){
        t = 0;
        for(int outer = 0; outer < sz; ++outer){
          t += tr[outer];
        }
        gradk[ikv][l][m] -= t;
      }
    }

    // // calc k 0 
    // for(int outer = 0; outer < d; ++outer){
    //   tv = 0;
    //   t = 0;
    //   for(int l = 0; l < nq; l++){
    //     tv += a1*q[l][i][m]*grad_output[l][i][outer];
    //     t += a1*o[l][i][outer]*q[l][i][m]*grad_output[l][i][outer];
    //   }
    //   for(int l = 0; l < nk; l++){
    //     gradk[l][i][m] += tv*v[l][i][outer] - t;
    //   }
    // }

    // // calc k 1
    // if(p == 2){
    //   for(int outer = 0; outer < d; ++outer){
    //     for(int rr = 0; rr < d/sz; ++rr){
    //       for(int r = 0; r < sz; ++r) tr[r]= 0;
    //       for(int l = 0; l < nq; l++){
    //         s[d+m] = q[l][i][m];
    //         tv = q[l][i][m]*grad_output[l][i][outer];
    //         __syncthreads();
    //         loc1 = d+rr*sz;
    //         for(int r = 0; r < sz; ++r){
    //           tr[r] += tv*s[loc1+r];
    //         }
    //       }
    //       for(int l = 0; l < nk; l++){
    //         s[m] = k[l][i][m];
    //         __syncthreads();
    //         loc2 = rr*sz;
    //         t = 0;
    //         for(int r = 0; r < sz; ++r){
    //           t += tr[r]*s[loc2+r];
    //         }
    //         gradk[l][i][m] += a22*t*v[l][i][outer];
    //       }

    //       for(int r = 0; r < sz; ++r) tr[r]= 0;
    //       for(int l = 0; l < nq; l++){
    //         s[d+m] = q[l][i][m];
    //         tv = o[l][i][outer]*q[l][i][m]*grad_output[l][i][outer];
    //         __syncthreads();
    //         loc1 = d+rr*sz;
    //         for(int r = 0; r < sz; ++r){
    //           tr[r] += tv*s[loc1+r];
    //         }
    //       }
    //       for(int l = 0; l < nk; l++){
    //         s[m] = k[l][i][m];
    //         __syncthreads();
    //         loc2 = rr*sz;
    //         t = 0;
    //         for(int r = 0; r < sz; ++r){
    //           t += tr[r]*s[loc2+r];
    //         }
    //         gradk[l][i][m] -= a22*t;
    //       }
    //     }
    //   }
    // }
  }
}

__global__
void calc_gradk_masked(torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> q, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> k, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> v, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> o, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> gradk, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> grad_output, int bh, int nq, int nk, int d, float a1, float a22, int p,int bhratio){

  extern __shared__ float s[];
  const int m = threadIdx.x;
  const int i = blockIdx.x;
  int ikv;
  float tv, t;
  int loc1, loc2;
  float tr[32];
  int sz = min(32,d);
  if(m < d && i < bh){
    ikv = i/bhratio;
    // MASKED PART ////////////////////////////
    // calc k 0 
    for(int oo = 0; oo < d/sz; ++oo){
      for(int outer = 0; outer < sz; ++outer) tr[outer] = 0;
      for(int l = nq-1; l >= 0; --l){
        t = 0;
        tv = q[i][l][m];
        s[m] = grad_output[i][l][m];
        s[d+m] = v[ikv][l][m];
        __syncthreads();
        for(int outer = 0; outer < sz; ++outer){
          int ooo = oo*sz + outer;
          tr[outer] += a1*tv*s[ooo];
          t += tr[outer]*s[d+ooo];
        }
        gradk[ikv][l][m] += t;
      }
      for(int l = nk-nq-1; l >= 0; --l){
        t = 0;
        s[d+m] = v[ikv][l][m];
        __syncthreads();
        for(int outer = 0; outer < sz; ++outer){
          t += tr[outer]*s[d+oo*sz+outer];
        }
        gradk[ikv][l][m] += t;
      }

      for(int outer = 0; outer < sz; ++outer) tr[outer] = 0;
      for(int l = nq-1; l >= 0; --l){
        t = 0;
        tv = q[i][l][m];
        s[m] = grad_output[i][l][m];
        s[d+m] = o[i][l][m];
        __syncthreads();
        for(int outer = 0; outer < sz; ++outer){
          int ooo = oo*sz + outer;
          tr[outer] += a1*s[d+ooo]*tv*s[ooo];
          t += tr[outer];
        }
        gradk[ikv][l][m] -= t;
      }
      for(int l = nk-nq-1; l >= 0; --l){
        t = 0;
        for(int outer = 0; outer < sz; ++outer){
          t += tr[outer];
        }
        gradk[ikv][l][m] -= t;
      }
    }

    // // calc k 0 
    // for(int outer = 0; outer < d; ++outer){
    //   tv = 0;
    //   t = 0;
    //   for(int l = nq-1; l >= 0; --l){
    //     tv += a1*q[l][i][m]*grad_output[l][i][outer];
    //     t += a1*o[l][i][outer]*q[l][i][m]*grad_output[l][i][outer];
    //     gradk[l][i][m] += tv*v[l][i][outer] - t;
    //   }
    //   for(int l = nk-nq-1; l >= 0; --l){
    //     gradk[l][i][m] += tv*v[l][i][outer] - t;
    //   }
    // }

    // // calc k 1
    // if(p == 2){
    //   for(int outer = 0; outer < d; ++outer){
    //     for(int rr = 0; rr < d/sz; ++rr){
    //       for(int r = 0; r < sz; ++r) tr[r]= 0;
    //       for(int l = nq-1; l >= 0; --l){
    //         s[d+m] = q[l][i][m];
    //         tv = q[l][i][m]*grad_output[l][i][outer];
    //         s[m] = k[l][i][m];
    //         __syncthreads();
    //         loc1 = d+rr*sz;
    //         for(int r = 0; r < sz; ++r){
    //           tr[r] += tv*s[loc1+r];
    //         }
    //         loc2 = rr*sz;
    //         t = 0;
    //         for(int r = 0; r < sz; ++r){
    //           t += tr[r]*s[loc2+r];
    //         }
    //         gradk[l][i][m] += a22*t*v[l][i][outer];
    //       }
    //       for(int l = nk-nq-1; l >= 0; --l){
    //         s[m] = k[l][i][m];
    //         __syncthreads();
    //         loc2 = rr*sz;
    //         t = 0;
    //         for(int r = 0; r < sz; ++r){
    //           t += tr[r]*s[loc2+r];
    //         }
    //         gradk[l][i][m] += a22*t*v[l][i][outer];
    //       }

    //       for(int r = 0; r < sz; ++r) tr[r]= 0;
    //       for(int l = nq-1; l >= 0; --l){
    //         s[d+m] = q[l][i][m];
    //         tv = o[l][i][outer]*q[l][i][m]*grad_output[l][i][outer];
    //         s[m] = k[l][i][m];
    //         __syncthreads();
    //         loc1 = d+rr*sz;
    //         for(int r = 0; r < sz; ++r){
    //           tr[r] += tv*s[loc1+r];
    //         }
    //         loc2 = rr*sz;
    //         t = 0;
    //         for(int r = 0; r < sz; ++r){
    //           t += tr[r]*s[loc2+r];
    //         }
    //         gradk[l][i][m] -= a22*t;
    //       }
    //       for(int l = nk-nq-1; l >= 0; --l){
    //         s[m] = k[l][i][m];
    //         __syncthreads();
    //         loc2 = rr*sz;
    //         t = 0;
    //         for(int r = 0; r < sz; ++r){
    //           t += tr[r]*s[loc2+r];
    //         }
    //         gradk[l][i][m] -= a22*t;
    //       }
    //     }
    //   }
    // }
  }
}

__global__
void calc_gradv_unmasked(torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> q, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> k, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> v, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> o, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> gradv, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> grad_output, int bh, int nq, int nk, int d, float a0, float a1, float a2, int p,int bhratio){

  extern __shared__ float s[];
  const int outer = threadIdx.x;
  const int i = blockIdx.x;
  int ikv;
  float tv, t;
  int loc1, loc2;
  float tr[32];
  int sz = min(32,d);
  if(i < bh && outer < d){
    ikv = i/bhratio;
    // UNMASKED PART ////////////////////////////
    // calc v 0    
    t = 0;
    for(int l = 0; l < nq; ++l){
      t += a0*grad_output[i][l][outer];
    }
    for(int l = 0; l < nk; ++l){
      gradv[ikv][l][outer] += t;
    }

    // calc v 1
    
    for(int mm = 0; mm < d/sz; ++mm){
      for(int m = 0; m < sz; ++m) tr[m] = 0;
      for(int l = 0; l < nq; ++l){
        s[outer] = q[i][l][outer];
        __syncthreads();
        for(int m = 0; m < sz; ++m){
          tr[m] += a1*s[mm*sz+m] * grad_output[i][l][outer];
        }
      }
      for(int l = 0; l < nk; ++l){
        t = 0;
        s[d+outer] = k[ikv][l][outer];
        __syncthreads();
        for(int m = 0; m < sz; ++m){
          t += tr[m]*s[d+mm*sz+m];
        }
        gradv[ikv][l][outer] += t;
      }
    }

    // // calc v 0
    // t = 0;
    // for(int l = 0; l < nq; ++l){
    //   t += a0*grad_output[i][l][outer];
    // }
    // for(int l = 0; l < nk; ++l){
    //   gradv[ikv][l][outer] += t;
    // }

    // // calc v 1
    // for(int m = 0; m < d; ++m){
    //   t = 0;
    //   for(int l = 0; l < nq; ++l){
    //     t += a1*q[i][l][m] * grad_output[i][l][outer];
    //   }
    //   for(int l = 0; l < nk; ++l){
    //     gradv[ikv][l][outer] += t*k[ikv][l][m];
    //   }
    // }

    // // calc v 2
    // if(p == 2){
    //   for(int m = 0; m < d; ++m){
    //     for(int rr = 0; rr < d/sz; ++rr){
    //       for(int r = 0; r < sz; ++r) tr[r]= 0;
    //       for(int l = 0; l < nq; l++){
    //         s[outer] = q[l][i][m]*q[l][i][outer];
    //         tv = grad_output[l][i][outer];
    //         __syncthreads();
    //         loc1 = rr*sz;
    //         for(int r = 0; r < sz; ++r){
    //           tr[r] += s[loc1+r]*tv;
    //         }
    //       }
    //       for(int l = 0; l < nk; l++){
    //         s[d+outer] = k[l][i][m]*k[l][i][outer];
    //         __syncthreads();
    //         loc2 = d+rr*sz;
    //         t = 0;
    //         for(int r = 0; r < sz; ++r){
    //           t += tr[r]*s[loc2+r];
    //         }
    //         gradv[l][i][outer] += a2*t;
    //       }
    //     }
    //   }
    // }
  }
}

__global__
void calc_gradv_masked(torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> q, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> k, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> v, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> o, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> gradv, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> grad_output, int bh, int nq, int nk, int d, float a0, float a1, float a2, int p,int bhratio){

  extern __shared__ float s[];
  const int outer = threadIdx.x;
  const int i = blockIdx.x;
  int ikv;
  float tv, t;
  int loc1, loc2;
  float tr[32];
  int sz = min(32,d);
  if(i < bh && outer < d){
    ikv = i/bhratio;
    // MASKED PART ////////////////////////////
    // calc v 0    
    t = 0;
    for(int l = nq-1; l >= 0; --l){
      t += a0*grad_output[i][l][outer];
      gradv[ikv][l][outer] += a0*t;
    }
    for(int l = nk-nq-1; l >= 0; --l){
      gradv[ikv][l][outer] += a0*t;
    }

    // calc v 1
    
    for(int mm = 0; mm < d/sz; ++mm){
      for(int m = 0; m < sz; ++m) tr[m] = 0;
      for(int l = nq-1; l >= 0; --l){
        t = 0;
        s[outer] = q[i][l][outer];
        s[d+outer] = k[ikv][l][outer];
        __syncthreads();
        for(int m = 0; m < sz; ++m){
          int mmm = mm*sz + m;
          tr[m] += a1*s[mmm] * grad_output[i][l][outer];
          t += tr[m]*s[d+mmm];
        }
        gradv[ikv][l][outer] += t;
      }
      for(int l = nk-nq-1; l >= 0; --l){
        t = 0;
        s[d+outer] = k[ikv][l][outer];
        __syncthreads();
        for(int m = 0; m < sz; ++m){
          t += tr[m]*s[d+mm*sz+m];
        }
        gradv[ikv][l][outer] += t;
      }
    }
    

    // // calc v 0    
    // t = 0;
    // for(int l = nq-1; l >= 0; --l){
    //   t += a0*grad_output[l][i][outer];
    //   gradv[l][i][outer] += a0*t;
    // }
    // for(int l = nk-nq-1; l >= 0; --l){
    //   gradv[l][i][outer] += a0*t;
    // }

    // // calc v 1
    // for(int m = 0; m < d; ++m){
    //   t = 0;
    //   for(int l = nq-1; l >= 0; --l){
    //     t += a1*q[l][i][m] * grad_output[l][i][outer];
    //     gradv[l][i][outer] += t*k[l][i][m];
    //   }
    //   for(int l = nk-nq-1; l >= 0; --l){
    //     gradv[l][i][outer] += t*k[l][i][m];
    //   }
    // }

    // // calc v 2
    // if(p == 2){
    //   for(int m = 0; m < d; ++m){
    //     for(int rr = 0; rr < d/sz; ++rr){
    //       for(int r = 0; r < sz; ++r) tr[r]= 0;
    //       for(int l = nq-1; l >= 0; --l){
    //         s[outer] = q[l][i][m]*q[l][i][outer];
    //         tv = grad_output[l][i][outer];
    //         s[d+outer] = k[l][i][m]*k[l][i][outer];
    //         __syncthreads();
    //         loc1 = rr*sz;
    //         for(int r = 0; r < sz; ++r){
    //           tr[r] += s[loc1+r]*tv;
    //         }
    //         loc2 = d+rr*sz;
    //         t = 0;
    //         for(int r = 0; r < sz; ++r){
    //           t += tr[r]*s[loc2+r];
    //         }
    //         gradv[l][i][outer] += a2*t;
    //       }
    //       for(int l = nk-nq-1; l >= 0; --l){
    //         s[d+outer] = k[l][i][m]*k[l][i][outer];
    //         __syncthreads();
    //         loc2 = d+rr*sz;
    //         t = 0;
    //         for(int r = 0; r < sz; ++r){
    //           t += tr[r]*s[loc2+r];
    //         }
    //         gradv[l][i][outer] += a2*t;
    //       }
    //     }
    //   }
    // }
  }
}


__global__
void div_grad_output(torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> grad_output, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> o, int bh, int nq, int d){
  const int m = threadIdx.x;
  const int i = blockIdx.x;
  if(m < d && i < bh){
    for(int l = 0; l < nq; ++l) grad_output[i][l][m] /= o[i][l][d];
  }
}

__global__
void apply_permute(torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> a, torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> a_p, int b, int h, int n, int d, int dir){
  const int m = threadIdx.x;
  const int j = blockIdx.x;
  const int i = blockIdx.y;
  if(m < d && i < b && j < h){
    for(int l = 0; l < n; ++l){
      // if(dir == 0) a_p[l][i*h+j][m] = a[i][l][j][m];
      // else a[i][l][j][m] = a_p[l][i*h+j][m];
      a[i][l][j][m] = a_p[l][i*h+j][m];
    }
  }
}

} // namespace

std::vector<torch::Tensor> backward_cuda(
    torch::Tensor q,
    torch::Tensor k,
    torch::Tensor v,
    torch::Tensor o,
    torch::Tensor grad_output,  
    // torch::Tensor q_old,
    // torch::Tensor k_old,
    // torch::Tensor v_old,
    // torch::Tensor o_old,
    // torch::Tensor grad_output_old,  
    bool mask,
    float a0,
    float a1,
    float a2,
    int p){

    // q: (nq,bh,d)
    // k: (nk,bh,d)
    // v: (nk,bh,d)
    // grad_output: (nq,bh,d)

    // gradq_coeffs0 = lin
    // gradq_coeffs1 = quad
    // gradq_coeffs0v = lin[:,:d,:]
    // gradq_coeffs0o = lin[:,d,:]
    // gradq_coeffs1v = quad[:,:d,:,:]
    // gradq_coeffs1o = quad[:,d,:,:]
    // o[:,d,:] = denum

  // const auto nq = q_old.size(1);
  // const auto nk = k_old.size(1);
  // const auto b = q_old.size(0);
  // const auto h = q_old.size(2);
  // const auto d = q_old.size(3);
  // const auto bh = b*h;

  const auto nq = q.size(1);
  const auto nk = k.size(1);
  const auto bh = q.size(0);
  const auto bhkv = k.size(0);
  const auto d = q.size(2);
  const int bhratio = bh/bhkv;

  const int threads = d;
  const int blocks = bh;
  
  auto opts =  torch::TensorOptions().dtype(torch::kFloat32).layout(torch::kStrided).device(torch::kCUDA, 0);
  // auto q = torch::zeros({nq,bh,d},opts);
  // auto k = torch::zeros({nk,bh,d},opts);
  // auto v = torch::zeros({nk,bh,d},opts);
  // auto o = torch::zeros({nq,bh,d+1},opts);
  // auto grad_output = torch::zeros({nq,bh,d},opts);
  auto gradq = torch::zeros({bh,nq,d},opts);
  auto gradk = torch::zeros({bh,nk,d},opts);
  auto gradv = torch::zeros({bh,nk,d},opts);
  // auto gradq_out = torch::zeros({b,nq,h,d},opts);
  // auto gradk_out = torch::zeros({b,nk,h,d},opts);
  // auto gradv_out = torch::zeros({b,nk,h,d},opts);

  // apply_permute<<<dim3(h,b),threads>>>(q_old.packed_accessor32<float,4,torch::RestrictPtrTraits>(),q.packed_accessor32<float,3,torch::RestrictPtrTraits>(),b,h,nq,d,0);
  // apply_permute<<<dim3(h,b),threads>>>(k_old.packed_accessor32<float,4,torch::RestrictPtrTraits>(),k.packed_accessor32<float,3,torch::RestrictPtrTraits>(),b,h,nk,d,0);
  // apply_permute<<<dim3(h,b),threads>>>(v_old.packed_accessor32<float,4,torch::RestrictPtrTraits>(),v.packed_accessor32<float,3,torch::RestrictPtrTraits>(),b,h,nk,d,0);
  // apply_permute<<<dim3(h,b),threads+1>>>(o_old.packed_accessor32<float,4,torch::RestrictPtrTraits>(),o.packed_accessor32<float,3,torch::RestrictPtrTraits>(),b,h,nq,d+1,0);
  // apply_permute<<<dim3(h,b),threads>>>(grad_output_old.packed_accessor32<float,4,torch::RestrictPtrTraits>(),grad_output.packed_accessor32<float,3,torch::RestrictPtrTraits>(),b,h,nq,d,0);

  div_grad_output<<<blocks,threads>>>(grad_output.packed_accessor32<float,3,torch::RestrictPtrTraits>(), o.packed_accessor32<float,3,torch::RestrictPtrTraits>(), bh,nq,d);
  hipDeviceSynchronize();

  if(mask){
    calc_gradq_masked<<<blocks,threads,2*(d)*sizeof(float)>>>(q.packed_accessor32<float,3,torch::RestrictPtrTraits>(), k.packed_accessor32<float,3,torch::RestrictPtrTraits>(), v.packed_accessor32<float,3,torch::RestrictPtrTraits>(), o.packed_accessor32<float,3,torch::RestrictPtrTraits>(), gradq.packed_accessor32<float,3,torch::RestrictPtrTraits>(), grad_output.packed_accessor32<float,3,torch::RestrictPtrTraits>(), bh,nq,nk,d,a1,2*a2,p,bhratio);
    calc_gradk_masked<<<blocks,threads,2*(d)*sizeof(float)>>>(q.packed_accessor32<float,3,torch::RestrictPtrTraits>(), k.packed_accessor32<float,3,torch::RestrictPtrTraits>(), v.packed_accessor32<float,3,torch::RestrictPtrTraits>(), o.packed_accessor32<float,3,torch::RestrictPtrTraits>(), gradk.packed_accessor32<float,3,torch::RestrictPtrTraits>(), grad_output.packed_accessor32<float,3,torch::RestrictPtrTraits>(), bh,nq,nk,d,a1,2*a2,p,bhratio);
    calc_gradv_masked<<<blocks,threads,2*(d)*sizeof(float)>>>(q.packed_accessor32<float,3,torch::RestrictPtrTraits>(), k.packed_accessor32<float,3,torch::RestrictPtrTraits>(), v.packed_accessor32<float,3,torch::RestrictPtrTraits>(), o.packed_accessor32<float,3,torch::RestrictPtrTraits>(), gradv.packed_accessor32<float,3,torch::RestrictPtrTraits>(), grad_output.packed_accessor32<float,3,torch::RestrictPtrTraits>(), bh,nq,nk,d,a0,a1,a2,p,bhratio);
  }
  else{
    calc_gradq_unmasked<<<blocks,threads,2*(d)*sizeof(float)>>>(q.packed_accessor32<float,3,torch::RestrictPtrTraits>(), k.packed_accessor32<float,3,torch::RestrictPtrTraits>(), v.packed_accessor32<float,3,torch::RestrictPtrTraits>(), o.packed_accessor32<float,3,torch::RestrictPtrTraits>(), gradq.packed_accessor32<float,3,torch::RestrictPtrTraits>(), grad_output.packed_accessor32<float,3,torch::RestrictPtrTraits>(), bh,nq,nk,d,a1,2*a2,p,bhratio);
    calc_gradk_unmasked<<<blocks,threads,2*(d)*sizeof(float)>>>(q.packed_accessor32<float,3,torch::RestrictPtrTraits>(), k.packed_accessor32<float,3,torch::RestrictPtrTraits>(), v.packed_accessor32<float,3,torch::RestrictPtrTraits>(), o.packed_accessor32<float,3,torch::RestrictPtrTraits>(), gradk.packed_accessor32<float,3,torch::RestrictPtrTraits>(), grad_output.packed_accessor32<float,3,torch::RestrictPtrTraits>(), bh,nq,nk,d,a1,2*a2,p,bhratio);
    calc_gradv_unmasked<<<blocks,threads,2*(d)*sizeof(float)>>>(q.packed_accessor32<float,3,torch::RestrictPtrTraits>(), k.packed_accessor32<float,3,torch::RestrictPtrTraits>(), v.packed_accessor32<float,3,torch::RestrictPtrTraits>(), o.packed_accessor32<float,3,torch::RestrictPtrTraits>(), gradv.packed_accessor32<float,3,torch::RestrictPtrTraits>(), grad_output.packed_accessor32<float,3,torch::RestrictPtrTraits>(), bh,nq,nk,d,a0,a1,a2,p,bhratio);
  }

  hipDeviceSynchronize();

  // apply_permute<<<dim3(h,b),threads>>>(gradq_out.packed_accessor32<float,4,torch::RestrictPtrTraits>(),gradq.packed_accessor32<float,3,torch::RestrictPtrTraits>(),b,h,nq,d,1);
  // apply_permute<<<dim3(h,b),threads>>>(gradk_out.packed_accessor32<float,4,torch::RestrictPtrTraits>(),gradk.packed_accessor32<float,3,torch::RestrictPtrTraits>(),b,h,nk,d,1);
  // apply_permute<<<dim3(h,b),threads>>>(gradv_out.packed_accessor32<float,4,torch::RestrictPtrTraits>(),gradv.packed_accessor32<float,3,torch::RestrictPtrTraits>(),b,h,nk,d,1);

  // delete q;
  // delete k;
  // delete v;
  // delete o;
  // delete grad_output;
  // delete gradq;
  // delete gradk;
  // delete gradv;

  return {gradq,gradk,gradv};
}

